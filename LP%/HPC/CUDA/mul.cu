#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

// CUDA kernel for matrix multiplication
__global__ void matrix_multiply(float *a, float *b, float *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < n && col < n) {
        for (int i = 0; i < n; ++i) {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main() {
    int n = 1024; // Matrix dimension
    size_t size = n * n * sizeof(float);

    float *a, *b, *c;
    float *d_a, *d_b, *d_c;

    // Allocate host memory
    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);

    // Initialize matrices
    for (int i = 0; i < n * n; ++i) {
        a[i] = i % n;
        b[i] = i % n;
    }

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Define execution configuration
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((n + threads.x - 1) / threads.x, (n + threads.y - 1) / threads.y);

    // Measure execution time
    hipEvent_t start, stop;
    float elapsed_time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matrix_multiply<<<blocks, threads>>>(d_a, d_b, d_c, n);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("Elapsed time for matrix multiplication: %.2f ms\n", elapsed_time);

    // Optionally print a few results
    for (int i = 0; i < 5; i++) {
        printf("c[%d] = %f\n", i, c[i]);
    }

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}

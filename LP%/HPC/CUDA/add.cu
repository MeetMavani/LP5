#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main() {
    int N = 1 << 20; // 1 million elements
    size_t size = N * sizeof(float);

    // Allocate memory on host
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // Allocate memory on device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Kernel launch
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < 10; i++)
        printf("C[%d] = %f\n", i, h_C[i]);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
